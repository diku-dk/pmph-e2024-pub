#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "helper.h"

__global__ void squareKernel(float* d_in, float *d_out) {
    const unsigned int lid = threadIdx.x;
    const unsigned int gid = blockIdx.x*blockDim.x + lid;
    d_out[gid] = d_in[gid]*d_in[gid];
}

int main(int argc, char** argv) {
    unsigned int N = 32;
    unsigned int mem_size = N*sizeof(float);

    // allocate host memory
    float* h_in  = (float*) malloc(mem_size);
    float* h_out = (float*) malloc(mem_size);

    // initialize the memory
    for(unsigned int i=0; i<N; ++i) {
        h_in[i] = (float)i;
    }

    // allocate device memory
    float* d_in;
    float* d_out;
    hipMalloc((void**)&d_in,  mem_size);
    hipMalloc((void**)&d_out, mem_size);

    // copy host memory to device
    hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);

    // execute the kernel
    squareKernel<<< 1, N>>>(d_in, d_out);

    // if you want to make sure that the kernel
    //   has finished execution you need to
    //   synchronize. E.g., this IS needed for
    //   runtime measurements. 
    //   Please note that the execution of multiple
    //   kernels in Cuda does not need explicit
    //   synchronizations (which are expensive).
    hipDeviceSynchronize();

    // check for errors
    gpuAssert( hipPeekAtLastError() );

    // copy result from ddevice to host
    hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);

    // print result
    for(unsigned int i=0; i<N; ++i) printf("%.6f\n", h_out[i]);

    // clean-up memory
    free(h_in);       free(h_out);
    hipFree(d_in);   hipFree(d_out);
}
