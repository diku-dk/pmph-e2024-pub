#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "helper.h"

#define GPU_RUNS 100

__global__ void mul2Kernel(float* X, float *Y) {
    const unsigned int gid = threadIdx.x;
    Y[gid] = 2 * X[gid];
}

int main(int argc, char** argv) {
    unsigned int N;
    
    { // reading the number of elements 
      if (argc != 2) { 
        printf("Num Args is: %d instead of 1. Exiting!\n", argc); 
        exit(1);
      }

      N = atoi(argv[1]);
      printf("N is: %d\n", N);

      const unsigned int maxN = 500000000;
      if(N > maxN) {
          printf("N is too big; maximal value is %d. Exiting!\n", maxN);
          exit(2);
      }
    }

    // use the first CUDA device:
    hipSetDevice(0);

    unsigned int mem_size = N*sizeof(float);

    // allocate host memory
    float* h_in  = (float*) malloc(mem_size);
    float* h_out = (float*) malloc(mem_size);

    // initialize the memory
    for(unsigned int i=0; i<N; ++i) {
        h_in[i] = (float)i;
    }

    // allocate device memory
    float* d_in;
    float* d_out;
    hipMalloc((void**)&d_in,  mem_size);
    hipMalloc((void**)&d_out, mem_size);

    // copy host memory to device
    hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);

    // a small number of dry runs
    for(int r = 0; r < 1; r++) {
        mul2Kernel<<< 1, N>>>(d_in, d_out);
    }
  
    { // execute the kernel a number of times;
      // to measure performance use a large N, e.g., 200000000,
      // and increase GPU_RUNS to 100 or more. 
    
        double elapsed; struct timeval t_start, t_end, t_diff;
        gettimeofday(&t_start, NULL);

        for(int r = 0; r < GPU_RUNS; r++) {
            mul2Kernel<<< 1, N>>>(d_in, d_out);
        }
        hipDeviceSynchronize();
        // ^ `hipDeviceSynchronize` is needed for runtime
        //     measurements, since CUDA kernels are executed
        //     asynchronously, i.e., the CPU does not wait
        //     for the kernel to finish.
        //   However, `hipDeviceSynchronize` is expensive
        //     so we need to amortize it across many runs;
        //     hence, when measuring performance use a big
        //     N and increase GPU_RUNS to 100 or more.
        //   Sure, it would be better by using CUDA events, but
        //     the current procedure is simple & works well enough.
        //   Please note that the execution of multiple
        //     kernels in Cuda executes correctly without such
        //     explicit synchronization; we need this only for
        //     runtime measurement.
        
        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (1.0 * (t_diff.tv_sec*1e6+t_diff.tv_usec)) / GPU_RUNS;
        double gigabytespersec = (2.0 * N * 4.0) / (elapsed * 1000.0);
        printf("The kernel took on average %f microseconds. GB/sec: %f \n", elapsed, gigabytespersec);
        
    }
        
    // check for errors
    gpuAssert( hipPeekAtLastError() );

    // copy result from ddevice to host
    hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);

    // print result
    //for(unsigned int i=0; i<N; ++i) printf("%.6f\n", h_out[i]);

    for(unsigned int i=0; i<N; ++i) {
        float actual   = h_out[i];
        float expected = 2 * h_in[i]; 
        if( actual != expected ) {
            printf("Invalid result at index %d, actual: %f, expected: %f. \n", i, actual, expected);
            exit(3);
        }
    }
    printf("Successful Validation.\n");

    // clean-up memory
    free(h_in);       free(h_out);
    hipFree(d_in);   hipFree(d_out);
}
