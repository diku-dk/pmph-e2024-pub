//#include "../../cub-1.8.0/hipcub/hipcub.hpp"   // or equivalently <cub/device/device_histogram.cuh>
#include "hipcub/hipcub.hpp"
#include "helper.cu.h"

template<class Z>
bool validateZ(Z* A, uint32_t sizeAB) {
    for(uint32_t i = 1; i < sizeAB; i++)
      if (A[i-1] > A[i]){
        printf("INVALID RESULT for i:%d, (A[i-1]=%d > A[i]=%d)\n", i, A[i-1], A[i]);
        return false;
      }
    return true;
}

void randomInitNat(uint32_t* data, const uint32_t size, const uint32_t H) {
    for (int i = 0; i < size; ++i) {
        unsigned long int r = rand();
        data[i] = r % H;
    }
}

double sortRedByKeyCUB( uint32_t* data_keys_in
                      , uint32_t* data_keys_out
                      , const uint64_t N
) {
    int beg_bit = 0;
    int end_bit = 32;

    void * tmp_sort_mem = NULL;
    size_t tmp_sort_len = 0;

    { // sort prelude
        hipcub::DeviceRadixSort::SortKeys( tmp_sort_mem, tmp_sort_len
                                      , data_keys_in, data_keys_out
                                      , N,   beg_bit,  end_bit
                                      );
        hipMalloc(&tmp_sort_mem, tmp_sort_len);
    }
    cudaCheckError();

    { // one dry run
        hipcub::DeviceRadixSort::SortKeys( tmp_sort_mem, tmp_sort_len
                                      , data_keys_in, data_keys_out
                                      , N,   beg_bit,  end_bit
                                      );
        hipDeviceSynchronize();
    }
    cudaCheckError();

    // timing
    double elapsed;
    struct timeval t_start, t_end, t_diff;
    gettimeofday(&t_start, NULL);

    for(int k=0; k<GPU_RUNS; k++) {
        hipcub::DeviceRadixSort::SortKeys( tmp_sort_mem, tmp_sort_len
                                      , data_keys_in, data_keys_out
                                      , N,   beg_bit,  end_bit
                                      );
    }
    hipDeviceSynchronize();
    cudaCheckError();

    gettimeofday(&t_end, NULL);
    timeval_subtract(&t_diff, &t_end, &t_start);
    elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / ((double)GPU_RUNS);

    hipFree(tmp_sort_mem);

    return elapsed;
}


int main (int argc, char * argv[]) {
    if (argc != 2) {
        printf("Usage: %s <size-of-array>\n", argv[0]);
        exit(1);
    }
    const uint64_t N = atoi(argv[1]);

    //Allocate and Initialize Host data with random values
    uint32_t* h_keys  = (uint32_t*) malloc(N*sizeof(uint32_t));
    uint32_t* h_keys_res  = (uint32_t*) malloc(N*sizeof(uint32_t));
    randomInitNat(h_keys, N, N/10);

    //Allocate and Initialize Device data
    uint32_t* d_keys_in;
    uint32_t* d_keys_out;
    cudaSucceeded(hipMalloc((void**) &d_keys_in,  N * sizeof(uint32_t)));
    cudaSucceeded(hipMemcpy(d_keys_in, h_keys, N * sizeof(uint32_t), hipMemcpyHostToDevice));
    cudaSucceeded(hipMalloc((void**) &d_keys_out, N * sizeof(uint32_t)));

    double elapsed = sortRedByKeyCUB( d_keys_in, d_keys_out, N );

    hipMemcpy(h_keys_res, d_keys_out, N*sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    cudaCheckError();

    bool success = validateZ(h_keys_res, N);

    printf("CUB Sorting for N=%lu runs in: %.2f us, VALID: %d\n", N, elapsed, success);

    // Cleanup and closing
    hipFree(d_keys_in); hipFree(d_keys_out);
    free(h_keys); free(h_keys_res);

    return success ? 0 : 1;
}
